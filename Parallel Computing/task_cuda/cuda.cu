#include "hip/hip_runtime.h"
#include <cmath>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>
#include <iostream>

__global__ void kernel(const float *__restrict__ A_ptr,
                       const float *__restrict__ B_ptr,
                       float *__restrict__ result_ptr,
                       const int num_elements) {
  // TODO: Implement the Kernel Logic
  // Sum up all the elements in the input tensor
  int index = blockIdx.x *blockDim.x+threadIdx.x;
  int stride = blockDim.x *gridDim.x;
  for(int i = index;i<num_elements;i += stride)  result_ptr[i] = A_ptr[i] + B_ptr[i];
}

static void launch_kernel(const void *A_ptr, const void *B_ptr, void *output_ptr,
                          const int num_elements) {
  // TODO: Implement the LaunchKernel Logic
  int blockSize = 32;
  int numBlocks = (num_elements + blockSize - 1)/blockSize;
  kernel<<<numBlocks,blockSize>>>((const float*)A_ptr,(const float*)B_ptr, (float*)output_ptr, num_elements);
  std::cout << "\n\nWarning: Need to implement the this!!\n\n" << std::endl;
  hipDeviceSynchronize();
}

//CUDA包装函数（暴露给py的）
torch::Tensor test_kernel(const torch::Tensor &A, const torch::Tensor &B) {
  torch::Tensor result_tensor = torch::empty_like(A);
  const int element_count = A.numel();

  launch_kernel(A.data_ptr<float>(), B.data_ptr<float>(), result_tensor.data_ptr<float>(),
                element_count);

  return result_tensor;
}

//此模块是pybind宏，将C++函数暴露给py环境使用
//CUDA_Test这个名字到时候被python import
//定义了py中名为"test_kernel"的函数，第三个参数是文档字符串
PYBIND11_MODULE(CUDA_Test, m) {
  m.def("test_kernel", &test_kernel, "Test kernel");
}