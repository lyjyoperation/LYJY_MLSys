//#include "cutlass/cutlass.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <cstdlib>
#include <cmath>

#define OFFSET(row, col, ld) ((row) * (ld) + (col))
#define FETCH_FLOAT4(pointer) (reinterpret_cast<float4*>(&(pointer))[0])

__global__ void gemm_baseline_kernel(float *A, float *B, float *C, int M, int N, int K) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < M && col < N) {
        float sum = 0.0f;
        for (int k = 0; k < K; k++) {
            sum += A[row * K + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}
template <
    const int BLOCK_SIZE_M,  // height of block of C that each thread block calculate
    const int BLOCK_SIZE_K,  // width of block of A that each thread block load into shared memory
    const int BLOCK_SIZE_N,  // width of block of C that each thread block calculate
    const int THREAD_SIZE_Y, // height of block of C that each thread calculate
    const int THREAD_SIZE_X,  // width of block of C that each thread calculate
    const bool ENABLE_DOUBLE_BUFFER // whether enable double buffering or not
    > 
__global__ void gemm_kernel( 
    float * __restrict__ A,
    float * __restrict__ B,
    float * __restrict__ C, 
    const int M,
    const int N,
    const int K) {
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    
    // the threads number in Block of X,Y
    const int THREAD_X_PER_BLOCK = BLOCK_SIZE_N / THREAD_SIZE_X;
    const int THREAD_Y_PER_BLOCK = BLOCK_SIZE_M / THREAD_SIZE_Y;
    const int THREAD_NUM_PER_BLOCK = THREAD_X_PER_BLOCK * THREAD_Y_PER_BLOCK;

    // thread id in cur Block
    const int tid = ty * THREAD_X_PER_BLOCK + tx;

    // shared memory
    __shared__ float As[2][BLOCK_SIZE_K][BLOCK_SIZE_M];
    __shared__ float Bs[2][BLOCK_SIZE_K][BLOCK_SIZE_N];
    // registers for C
    float accum[THREAD_SIZE_Y][THREAD_SIZE_X];
    #pragma unroll
    for(int i=0; i<THREAD_SIZE_Y; i++){
        #pragma unroll
        for(int j=0; j<THREAD_SIZE_X; j++){
            accum[i][j]=0.0;
        }
    }
    // registers for A and B
    float frag_a[2][THREAD_SIZE_Y];
    float frag_b[2][THREAD_SIZE_X];
    // registers load global memory
    const int ldg_num_a = BLOCK_SIZE_M * BLOCK_SIZE_K / (THREAD_NUM_PER_BLOCK * 4);
    const int ldg_num_b = BLOCK_SIZE_K * BLOCK_SIZE_N / (THREAD_NUM_PER_BLOCK * 4);
    float ldg_a_reg[4*ldg_num_a];
    float ldg_b_reg[4*ldg_num_b];

    // threads number in one row
    const int A_TILE_THREAD_PER_ROW = BLOCK_SIZE_K / 4;
    const int B_TILE_THREAD_PER_ROW = BLOCK_SIZE_N / 4;

    // row number and col number that needs to be loaded by this thread
    const int A_TILE_ROW_START = tid / A_TILE_THREAD_PER_ROW;
    const int B_TILE_ROW_START = tid / B_TILE_THREAD_PER_ROW;

    const int A_TILE_COL = tid % A_TILE_THREAD_PER_ROW * 4; 
    const int B_TILE_COL = tid % B_TILE_THREAD_PER_ROW * 4;

    // row stride that thread uses to load multiple rows of a tile
    const int A_TILE_ROW_STRIDE = THREAD_NUM_PER_BLOCK / A_TILE_THREAD_PER_ROW;
    const int B_TILE_ROW_STRIDE = THREAD_NUM_PER_BLOCK / B_TILE_THREAD_PER_ROW;
    //将指针重新定位到当前线程块需要处理的矩阵块的起始位置
    A = &A[(BLOCK_SIZE_M * by)* K];
    B = &B[BLOCK_SIZE_N * bx];

    //一个wrap32个线程，tid/32确定是哪一个线程里面的，%32是在wrap中哪个位置
    const int warp_id = tid / 32;
    const int lane_id = tid % 32;
    const int a_tile_index =  warp_id/2*16 + lane_id/8*4; //warp_id * 8 + (lane_id / 16)*4; // (warp_id/4)*32 + ((lane_id%16)/2)*4;
    const int b_tile_index =  warp_id%2*32 + lane_id%8*4; //(lane_id % 16) * 4; // (warp_id%4)*16 + (lane_id/16)*8 + (lane_id%2)*4;
    
    //transfer first tile from global mem to shared mem
    // load A from global memory to shared memory
    #pragma unroll
    for ( int i = 0 ; i < BLOCK_SIZE_M ; i += A_TILE_ROW_STRIDE) {
        int ldg_index = i / A_TILE_ROW_STRIDE * 4;
        FETCH_FLOAT4(ldg_a_reg[ldg_index]) = FETCH_FLOAT4(A[OFFSET(
            A_TILE_ROW_START + i, // row
            A_TILE_COL, // col
            K )]);
        As[0][A_TILE_COL][A_TILE_ROW_START + i]=ldg_a_reg[ldg_index];
        As[0][A_TILE_COL+1][A_TILE_ROW_START + i]=ldg_a_reg[ldg_index+1];
        As[0][A_TILE_COL+2][A_TILE_ROW_START + i]=ldg_a_reg[ldg_index+2];
        As[0][A_TILE_COL+3][A_TILE_ROW_START + i]=ldg_a_reg[ldg_index+3];
    }
    // load B from global memory to shared memory
    #pragma unroll
    for ( int i = 0 ; i < BLOCK_SIZE_K; i += B_TILE_ROW_STRIDE) {
        FETCH_FLOAT4(Bs[0][B_TILE_ROW_START + i][B_TILE_COL]) = FETCH_FLOAT4(B[OFFSET(
                B_TILE_ROW_START + i, // row
                B_TILE_COL, // col
                N )]);
    }
    __syncthreads();
    
    // load A from shared memory to register
    FETCH_FLOAT4(frag_a[0][0]) = FETCH_FLOAT4(As[0][0][a_tile_index]);
    FETCH_FLOAT4(frag_a[0][4]) = FETCH_FLOAT4(As[0][0][a_tile_index + 64]);
    
    // load B from shared memory to register
    FETCH_FLOAT4(frag_b[0][0]) = FETCH_FLOAT4(Bs[0][0][b_tile_index]);
    FETCH_FLOAT4(frag_b[0][4]) = FETCH_FLOAT4(Bs[0][0][b_tile_index + 64]);
    
    int write_stage_idx = 1;
    int tile_idx = 0;
    do{
        // next tile index
        tile_idx += BLOCK_SIZE_K;
        // load next tile from global mem
        if(tile_idx< K){
            #pragma unroll
            for ( int i = 0 ; i < BLOCK_SIZE_M ; i += A_TILE_ROW_STRIDE) {
                int ldg_index = i / A_TILE_ROW_STRIDE * 4;
                FETCH_FLOAT4(ldg_a_reg[ldg_index]) = FETCH_FLOAT4(A[OFFSET(
                    A_TILE_ROW_START + i, // row
                    A_TILE_COL + tile_idx, // col
                    K )]);
            }
            #pragma unroll
            for ( int i = 0 ; i < BLOCK_SIZE_K; i += B_TILE_ROW_STRIDE) {
                int ldg_index = i / B_TILE_ROW_STRIDE * 4;
                FETCH_FLOAT4(ldg_b_reg[ldg_index]) = FETCH_FLOAT4(B[OFFSET(
                    tile_idx + B_TILE_ROW_START + i, // row
                    B_TILE_COL, // col
                    N )]);
            }
        }

        int load_stage_idx = write_stage_idx ^ 1;

        #pragma unroll
        for(int j=0; j<BLOCK_SIZE_K - 1; ++j){
            // load next tile from shared mem to register 
            // load A from shared memory to register
            FETCH_FLOAT4(frag_a[(j+1)%2][0]) = FETCH_FLOAT4(As[load_stage_idx][(j+1)][a_tile_index]);
            FETCH_FLOAT4(frag_a[(j+1)%2][4]) = FETCH_FLOAT4(As[load_stage_idx][(j+1)][a_tile_index + 64]);
            // load B from shared memory to register
            FETCH_FLOAT4(frag_b[(j+1)%2][0]) = FETCH_FLOAT4(Bs[load_stage_idx][(j+1)][b_tile_index]);
            FETCH_FLOAT4(frag_b[(j+1)%2][4]) = FETCH_FLOAT4(Bs[load_stage_idx][(j+1)][b_tile_index + 64]);
            // compute C THREAD_SIZE_X x THREAD_SIZE_Y
            #pragma unroll
            for (int thread_y = 0; thread_y < THREAD_SIZE_Y; ++thread_y) {
                #pragma unroll
                for (int thread_x = 0; thread_x < THREAD_SIZE_X; ++thread_x) {
                    accum[thread_y][thread_x] += frag_a[j%2][thread_y] * frag_b[j%2][thread_x];
                }
            }
        }

        if(tile_idx < K){
            // load A from global memory to shared memory
            #pragma unroll
            for ( int i = 0 ; i < BLOCK_SIZE_M ; i += A_TILE_ROW_STRIDE) {
                int ldg_index = i / A_TILE_ROW_STRIDE * 4;
                As[write_stage_idx][A_TILE_COL][A_TILE_ROW_START + i]=ldg_a_reg[ldg_index];
                As[write_stage_idx][A_TILE_COL+1][A_TILE_ROW_START + i]=ldg_a_reg[ldg_index+1];
                As[write_stage_idx][A_TILE_COL+2][A_TILE_ROW_START + i]=ldg_a_reg[ldg_index+2];
                As[write_stage_idx][A_TILE_COL+3][A_TILE_ROW_START + i]=ldg_a_reg[ldg_index+3];
            }
            // load B from global memory to shared memory
            #pragma unroll
            for ( int i = 0 ; i < BLOCK_SIZE_K; i += B_TILE_ROW_STRIDE) {
                int ldg_index = i / B_TILE_ROW_STRIDE * 4;
                FETCH_FLOAT4(Bs[write_stage_idx][B_TILE_ROW_START + i][B_TILE_COL]) = FETCH_FLOAT4(ldg_b_reg[ldg_index]);
            }
            // use double buffer, only need one sync
            __syncthreads();
            // switch
            write_stage_idx ^= 1;
        }

        // load first tile from shared mem to register of next iter
        // load A from shared memory to register
        FETCH_FLOAT4(frag_a[0][0]) = FETCH_FLOAT4(As[load_stage_idx^1][0][a_tile_index]);
        FETCH_FLOAT4(frag_a[0][4]) = FETCH_FLOAT4(As[load_stage_idx^1][0][a_tile_index + 64]);
        // load B from shared memory to register
        FETCH_FLOAT4(frag_b[0][0]) = FETCH_FLOAT4(Bs[load_stage_idx^1][0][b_tile_index]);
        FETCH_FLOAT4(frag_b[0][4]) = FETCH_FLOAT4(Bs[load_stage_idx^1][0][b_tile_index + 64]);
        // compute C THREAD_SIZE_X x THREAD_SIZE_Y
        #pragma unroll
        for (int thread_y = 0; thread_y < THREAD_SIZE_Y; ++thread_y) {
            #pragma unroll
            for (int thread_x = 0; thread_x < THREAD_SIZE_X; ++thread_x) {
                accum[thread_y][thread_x] += frag_a[1][thread_y] * frag_b[1][thread_x];
            }
        }
    }while(tile_idx< K);
    
    const int c_block_row = a_tile_index;
    const int c_block_col = b_tile_index;

    //store C00 block
    for(int i=0; i<4; i++){
      FETCH_FLOAT4(C[OFFSET(
        BLOCK_SIZE_M * by + c_block_row + i,
        BLOCK_SIZE_N * bx + c_block_col,
        N)]) = FETCH_FLOAT4(accum[i][0]);
    }
    //store C01 block
    for(int i=0; i<4; i++){
      FETCH_FLOAT4(C[OFFSET(
        BLOCK_SIZE_M * by + c_block_row + i,
        BLOCK_SIZE_N * bx + c_block_col + 64,
        N)]) = FETCH_FLOAT4(accum[i][4]);
    }
    //store C10 block
    for(int i=0; i<4; i++){
      FETCH_FLOAT4(C[OFFSET(
        BLOCK_SIZE_M * by + c_block_row + 64 + i,
        BLOCK_SIZE_N * bx + c_block_col,
        N)]) = FETCH_FLOAT4(accum[i+4][0]);
    }
    //store C11 block
    for(int i=0; i<4; i++){
      FETCH_FLOAT4(C[OFFSET(
        BLOCK_SIZE_M * by + c_block_row + 64 + i,
        BLOCK_SIZE_N * bx + c_block_col + 64,
        N)]) = FETCH_FLOAT4(accum[i+4][4]);
    }
}

void init_matrix(float *matrix, int size) {
    for (int i = 0; i < size; i++) {
        matrix[i] = static_cast<float>(rand()) / RAND_MAX * 2.0f - 1.0f; // [-1, 1]
    }
}

void cpu_gemm_reference(float *A, float *B, float *C_ref, int M, int N, int K) {
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            float sum = 0.0f;
            for (int k = 0; k < K; k++) {
                sum += A[i * K + k] * B[k * N + j];
            }
            C_ref[i * N + j] = sum;
        }
    }
}

bool verify_result(float *C1, float *C2, int size, float epsilon = 1e-4f) {
    for (int i = 0; i < size; i++) {
        if (std::abs(C1[i] - C2[i]) > epsilon) {
            std::cout << "Failed at " << i << ", GPU=" << C1[i] 
                     << ", CPU=" << C2[i] << ", diff=" << std::abs(C1[i] - C2[i]) << std::endl;
            return false;
        }
    }
    return true;
}

void test_cutlass() {
    int deviceCount = 0;
    hipError_t error = hipGetDeviceCount(&deviceCount);

    if (error != hipSuccess || deviceCount == 0) {
        std::cout << "No CUDA device found" << std::endl;
    } else {
        std::cout << "Found " << deviceCount << " CUDA devices" << std::endl;
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, 0);
        std::cout << "Device name: " << prop.name << std::endl;
        std::cout << "Compute capability: " << prop.major << "." << prop.minor << std::endl;
    }
}

float GEMM_Baseline(float *A, float *B, float *C, int M, int N, int K) {
    // 设备指针
    float *d_A, *d_B, *d_C;
    
    size_t size_A = M * K * sizeof(float);
    size_t size_B = K * N * sizeof(float);
    size_t size_C = M * N * sizeof(float);
    
    // 分配GPU内存
    hipMalloc(&d_A, size_A);
    hipMalloc(&d_B, size_B);
    hipMalloc(&d_C, size_C);
    
    // 将数据从主机复制到设备
    hipMemcpy(d_A, A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size_B, hipMemcpyHostToDevice);
    
    // 设置线程块和网格大小
    dim3 blockSize(16, 16);  // 16x16 线程块
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x, 
                  (M + blockSize.y - 1) / blockSize.y);
    
    // 创建CUDA事件用于计时
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    // 记录开始时间
    hipEventRecord(start);
    
    // 启动kernel
    gemm_baseline_kernel<<<gridSize, blockSize>>>(d_A, d_B, d_C, M, N, K);
    
    // 记录结束时间
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    // 计算执行时间
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    
    // 将结果复制回主机
    hipMemcpy(C, d_C, size_C, hipMemcpyDeviceToHost);
    
    // 计算性能指标
    double flops = 2.0 * M * N * K; // 每次乘加算2次浮点运算
    double gflops = flops / (milliseconds * 1e6); // 转换为GFLOPS
    
    std::cout << "Baseline GEMM [" << M << "x" << K << "] x [" << K << "x" << N 
              << "] = [" << M << "x" << N << "]" << std::endl;
    std::cout << "执行时间: " << milliseconds << " ms" << std::endl;
    std::cout << "性能: " << gflops << " GFLOPS" << std::endl;
    
    // 清理GPU内存
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    
    // 清理事件
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return milliseconds;
}

float GEMM_Kernel(float *A, float *B, float *C, int M, int N, int K) {
    // 设备指针
    float *d_A, *d_B, *d_C;
    
    size_t size_A = M * K * sizeof(float);
    size_t size_B = K * N * sizeof(float);
    size_t size_C = M * N * sizeof(float);
    
    // 分配GPU内存
    hipMalloc(&d_A, size_A);
    hipMalloc(&d_B, size_B);
    hipMalloc(&d_C, size_C);
    
    // 将数据从主机复制到设备
    hipMemcpy(d_A, A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size_B, hipMemcpyHostToDevice);
    
    const int BLOCK_SIZE_M = 128;
    const int BLOCK_SIZE_K = 8;
    const int BLOCK_SIZE_N = 128;
    const int THREAD_SIZE_X = 8;
    const int THREAD_SIZE_Y = 8;

    dim3 blockSize(BLOCK_SIZE_N / THREAD_SIZE_X, BLOCK_SIZE_M / THREAD_SIZE_Y);
    dim3 gridSize(N/BLOCK_SIZE_N, M/BLOCK_SIZE_M);
    
    // 创建CUDA事件用于计时
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    // 记录开始时间
    hipEventRecord(start);
    
    // 启动kernel
    gemm_kernel<BLOCK_SIZE_M, BLOCK_SIZE_K, BLOCK_SIZE_N, THREAD_SIZE_Y, THREAD_SIZE_X,true>
        <<<gridSize, blockSize>>>(d_A, d_B, d_C, M, N, K);
    
    // 记录结束时间
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    // 计算执行时间
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    
    // 将结果复制回主机
    hipMemcpy(C, d_C, size_C, hipMemcpyDeviceToHost);
    
    // 计算性能指标
    double flops = 2.0 * M * N * K; // 每次乘加算2次浮点运算
    double gflops = flops / (milliseconds * 1e6); // 转换为GFLOPS
    
    std::cout << "Opt GEMM [" << M << "x" << K << "] x [" << K << "x" << N 
              << "] = [" << M << "x" << N << "]" << std::endl;
    std::cout << "执行时间: " << milliseconds << " ms" << std::endl;
    std::cout << "性能: " << gflops << " GFLOPS" << std::endl;
    
    // 清理GPU内存
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    
    // 清理事件
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return milliseconds;
}

int main() {
    test_cutlass();
    srand(42);
    int test_sizes[] = {256, 512, 1024,2048};
    //int test_sizes[] = {4096};
    int num_tests = sizeof(test_sizes) / sizeof(test_sizes[0]);
    
    for (int t = 0; t < num_tests; t++) {
        int M = test_sizes[t];
        int N = test_sizes[t];
        int K = test_sizes[t];
        
        std::cout << "\n--- 测试矩阵大小: " << M << "x" << K << " x " 
                  << K << "x" << N << " ---" << std::endl;
        
        // 分配主机内存
        float *A = new float[M * K];
        float *B = new float[K * N];
        float *C = new float[M * N];
        float *COpt = new float[M*N];
        float *C_ref = new float[M * N];
        
        // 初始化输入矩阵
        init_matrix(A, M * K);
        init_matrix(B, K * N);
        
        // CPU参考实现（用于验证正确性）
        auto cpu_start = std::chrono::high_resolution_clock::now();
        cpu_gemm_reference(A, B, C_ref, M, N, K);
        auto cpu_end = std::chrono::high_resolution_clock::now();
        auto cpu_duration = std::chrono::duration_cast<std::chrono::milliseconds>(cpu_end - cpu_start);

        std::cout << "CPU参考时间: " << cpu_duration.count() << " ms" << std::endl;
        
        // CUDA Baseline实现
        float m1 = GEMM_Baseline(A, B, C, M, N, K);

        //CUDA优化版本实现
        float m2 = GEMM_Kernel(A, B, COpt, M, N, K);

        // 验证结果正确性,correct1用来验证CPU和GPU，correct2用来验证GPU和GPU优化
        bool correct1 = verify_result(C, C_ref, M * N);
        std::cout << "结果验证CPU和GPU: " << (correct1 ? "\033[1;32m通过\033[0m" : "\033[1;31m失败\033[0m") << std::endl;
        
        if (!correct1) {
            std::cout << "\033[1;31m警告: 结果不正确, 请检查实现! \033[0m" << std::endl;
        }

        bool correct2 = verify_result(COpt, C, M * N);
        std::cout << "结果验证GPU和GPUOPT: " << (correct2 ? "\033[1;32m通过\033[0m" : "\033[1;31m失败\033[0m") << std::endl;
        
        if (!correct2) {
            std::cout << "\033[1;31m警告: 结果不正确, 请检查实现! \033[0m" << std::endl;
        }

         //CUDA Baseline & CUDA OPT加速比的比较
        std::cout << "加速比 = "<< m1/m2 <<std::endl;

        // 释放内存
        delete[] A;
        delete[] B;
        delete[] C;
        delete[] COpt;
        delete[] C_ref;
    }
    
    std::cout << "\n=== 测试完成 ===" << std::endl;
   //std::cout << "提示：这是baseline实现，后续优化版本应该与此baseline进行对比" << std::endl;
    //std::cout << "加速比 = baseline时间 / 优化版本时间" << std::endl;
	// TODO: 实现你自己的Kernel，并计算和 Baseline 的加速比
   
    return 0;
}
